
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
using namespace std;
#define TPB 256
#define ARRAY_SIZE 10
#define N (ARRAY_SIZE/TPB + 1)


 __global__ void saxpy(float *x, float *y, const float a)
 {

   const int i = blockIdx.x*blockDim.x + threadIdx.x;

   if (i<ARRAY_SIZE) {
      y[i] = a*x[i] + y[i];
   }
}

int main()
{

  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(ARRAY_SIZE*sizeof(float));
  y = (float*)malloc(ARRAY_SIZE*sizeof(float));
  const int a = 3;

  hipMalloc(&d_x, ARRAY_SIZE*sizeof(float));
  hipMalloc(&d_y, ARRAY_SIZE*sizeof(float));

  for (int i = 0; i < ARRAY_SIZE; i++) {
    x[i] = rand() % 1000;
    y[i] = rand() % 1000;
        cout << x[i] << "\n";
            cout << y[i] << "\n\n";
  }
  cout << "---------------------" <<"\n";

  hipMemcpy(d_x, x, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);

  saxpy<<<N, TPB>>>(d_x, d_y, a);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < ARRAY_SIZE; i++) {
    cout << x[i] << "\n";
    cout << y[i] << "\n\n";
  }

  free(x);
  free(y);
  hipFree(d_x);
  hipFree(d_y);

  return 0;
}
