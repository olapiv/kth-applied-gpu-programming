#include "hip/hip_runtime.h"
#include "Particles.h"
#include "Alloc.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#define TpBx 512

/** allocate particle arrays */
void particle_allocate(struct parameters* param, struct particles* part, int is)
{
    
    // set species ID
    part->species_ID = is;
    // number of particles
    part->nop = param->np[is];
    // maximum number of particles
    part->npmax = param->npMax[is];
    
    // choose a different number of mover iterations for ions and electrons
    if (param->qom[is] < 0){  //electrons
        part->NiterMover = param->NiterMover;
        part->n_sub_cycles = param->n_sub_cycles;
    } else {                  // ions: only one iteration
        part->NiterMover = 1;
        part->n_sub_cycles = 1;
    }
    
    // particles per cell
    part->npcelx = param->npcelx[is];
    part->npcely = param->npcely[is];
    part->npcelz = param->npcelz[is];
    part->npcel = part->npcelx*part->npcely*part->npcelz;
    
    // cast it to required precision
    part->qom = (FPpart) param->qom[is];
    
    long npmax = part->npmax;
    
    // initialize drift and thermal velocities
    // drift
    part->u0 = (FPpart) param->u0[is];
    part->v0 = (FPpart) param->v0[is];
    part->w0 = (FPpart) param->w0[is];
    // thermal
    part->uth = (FPpart) param->uth[is];
    part->vth = (FPpart) param->vth[is];
    part->wth = (FPpart) param->wth[is];
    
    
    //////////////////////////////
    /// ALLOCATION PARTICLE ARRAYS
    //////////////////////////////
    part->x = new FPpart[npmax];
    part->y = new FPpart[npmax];
    part->z = new FPpart[npmax];
    // allocate velocity
    part->u = new FPpart[npmax];
    part->v = new FPpart[npmax];
    part->w = new FPpart[npmax];
    // allocate charge = q * statistical weight
    part->q = new FPinterp[npmax];
    
}
/** deallocate */
void particle_deallocate(struct particles* part)
{
    // deallocate particle variables
    delete[] part->x;
    delete[] part->y;
    delete[] part->z;
    delete[] part->u;
    delete[] part->v;
    delete[] part->w;
    delete[] part->q;
}

/** allocate particle arrays */
void particle_allocate_gpu(struct particles* part, struct particles* particlesGPU)
{    
    FPpart *dev_x, *dev_y, *dev_z, *dev_u, *dev_v, *dev_w, *dev_q;

    hipMalloc(&dev_x, part->npmax * sizeof(FPpart));
    hipMalloc(&dev_y, part->npmax * sizeof(FPpart));
    hipMalloc(&dev_z, part->npmax * sizeof(FPpart));
    hipMalloc(&dev_u, part->npmax * sizeof(FPpart));
    hipMalloc(&dev_v, part->npmax * sizeof(FPpart));
    hipMalloc(&dev_w, part->npmax * sizeof(FPpart));
    hipMalloc(&dev_q, part->npmax * sizeof(FPpart));

    hipMemcpy(dev_x, part->x, part->npmax * sizeof(*dev_x), hipMemcpyHostToDevice);
    hipMemcpy(dev_y, part->y, part->npmax * sizeof(*dev_y), hipMemcpyHostToDevice);
    hipMemcpy(dev_z, part->z, part->npmax * sizeof(*dev_z), hipMemcpyHostToDevice);
    hipMemcpy(dev_x, part->u, part->npmax * sizeof(*dev_u), hipMemcpyHostToDevice);
    hipMemcpy(dev_x, part->v, part->npmax * sizeof(*dev_v), hipMemcpyHostToDevice);
    hipMemcpy(dev_x, part->w, part->npmax * sizeof(*dev_w), hipMemcpyHostToDevice);
    hipMemcpy(dev_q, part->q, part->npmax * sizeof(*dev_q), hipMemcpyHostToDevice);

    // Binding pointers
    hipMemcpy(&(particlesGPU->x), &dev_x, sizeof(particlesGPU->x), hipMemcpyHostToDevice);
    hipMemcpy(&(particlesGPU->y), &dev_y, sizeof(particlesGPU->y), hipMemcpyHostToDevice);
    hipMemcpy(&(particlesGPU->z), &dev_z, sizeof(particlesGPU->z), hipMemcpyHostToDevice);
    hipMemcpy(&(particlesGPU->u), &dev_u, sizeof(particlesGPU->u), hipMemcpyHostToDevice);
    hipMemcpy(&(particlesGPU->v), &dev_v, sizeof(particlesGPU->v), hipMemcpyHostToDevice);
    hipMemcpy(&(particlesGPU->w), &dev_w, sizeof(particlesGPU->w), hipMemcpyHostToDevice);
    hipMemcpy(&(particlesGPU->q), &dev_q, sizeof(particlesGPU->q), hipMemcpyHostToDevice);
}

/** allocated interpolated densities per species */
void particle_deallocate_gpu(struct particles* part)
{
    hipFree(part->x);
    hipFree(part->y);
    hipFree(part->z);
    hipFree(part->u);
    hipFree(part->v);
    hipFree(part->w);
    hipFree(part->q);
}

__device__ void subcycle_single_particle(particles* part, EMfield* field, grid* grd, parameters* param, int particle_index) {

    // auxiliary variables
    FPpart dt_sub_cycling = (FPpart) param->dt/((double) part->n_sub_cycles);
    FPpart dto2 = .5*dt_sub_cycling, qomdt2 = part->qom*dto2/param->c;
    FPpart omdtsq, denom, ut, vt, wt, udotb;
    
    // local (to the particle) electric and magnetic field
    FPfield Exl=0.0, Eyl=0.0, Ezl=0.0, Bxl=0.0, Byl=0.0, Bzl=0.0;
    
    // interpolation densities
    int ix,iy,iz;
    FPfield weight[2][2][2];
    FPfield xi[2], eta[2], zeta[2];
    
    // intermediate particle position and velocity
    FPpart xptilde, yptilde, zptilde, uptilde, vptilde, wptilde;

    xptilde = part->x[particle_index];
    yptilde = part->y[particle_index];
    zptilde = part->z[particle_index];
    // calculate the average velocity iteratively
    for(int innter=0; innter < part->NiterMover; innter++){
        // interpolation G-->P
        // 2 + to create boundary conditions
        // Index of the cells:
        ix = 2 +  int((part->x[particle_index] - grd->xStart)*grd->invdx);
        iy = 2 +  int((part->y[particle_index] - grd->yStart)*grd->invdy);
        iz = 2 +  int((part->z[particle_index] - grd->zStart)*grd->invdz);
        
        // calculate weights
        long xi0_index_flat = get_idx(ix - 1, iy, iz, grd->nyn, grd->nzn);
        xi[0]   = part->x[particle_index] - grd->XN_flat[xi0_index_flat];

        long eta0_index_flat = get_idx(ix, iy - 1, iz, grd->nyn, grd->nzn);
        eta[0]  = part->y[particle_index] - grd->YN_flat[eta0_index_flat];

        long zeta0_index_flat = get_idx(ix, iy, iz - 1, grd->nyn, grd->nzn);
        zeta[0] = part->z[particle_index] - grd->ZN_flat[zeta0_index_flat];

        long index_flat_1 = get_idx(ix, iy, iz, grd->nyn, grd->nzn);
        xi[1]   = grd->XN_flat[index_flat_1] - part->x[particle_index];
        eta[1]  = grd->YN_flat[index_flat_1] - part->y[particle_index];
        zeta[1] = grd->ZN_flat[index_flat_1] - part->z[particle_index];

        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    weight[ii][jj][kk] = xi[ii] * eta[jj] * zeta[kk] * grd->invVOL;
        
        // set to zero local electric and magnetic field
        Exl=0.0, Eyl = 0.0, Ezl = 0.0, Bxl = 0.0, Byl = 0.0, Bzl = 0.0;
        
        for (int ii=0; ii < 2; ii++)
            for (int jj=0; jj < 2; jj++)
                for(int kk=0; kk < 2; kk++){
                    long index_flat = get_idx(ix- ii, iy -jj, iz- kk, grd->nyn, grd->nzn);
                    Exl += weight[ii][jj][kk]*field->Ex_flat[index_flat];
                    Eyl += weight[ii][jj][kk]*field->Ey_flat[index_flat];
                    Ezl += weight[ii][jj][kk]*field->Ez_flat[index_flat];
                    Bxl += weight[ii][jj][kk]*field->Bxn_flat[index_flat];
                    Byl += weight[ii][jj][kk]*field->Byn_flat[index_flat];
                    Bzl += weight[ii][jj][kk]*field->Bzn_flat[index_flat];
                }
        
        // end interpolation
        omdtsq = qomdt2*qomdt2*(Bxl*Bxl+Byl*Byl+Bzl*Bzl);
        denom = 1.0/(1.0 + omdtsq);
        // solve the position equation
        ut= part->u[particle_index] + qomdt2*Exl;
        vt= part->v[particle_index] + qomdt2*Eyl;
        wt= part->w[particle_index] + qomdt2*Ezl;
        udotb = ut*Bxl + vt*Byl + wt*Bzl;
        // solve the velocity equation
        uptilde = (ut+qomdt2*(vt*Bzl -wt*Byl + qomdt2*udotb*Bxl))*denom;
        vptilde = (vt+qomdt2*(wt*Bxl -ut*Bzl + qomdt2*udotb*Byl))*denom;
        wptilde = (wt+qomdt2*(ut*Byl -vt*Bxl + qomdt2*udotb*Bzl))*denom;
        // update position
        part->x[particle_index] = xptilde + uptilde*dto2;
        part->y[particle_index] = yptilde + vptilde*dto2;
        part->z[particle_index] = zptilde + wptilde*dto2;
        
        
    } // end of iteration
    // update the final position and velocity
    part->u[particle_index]= 2.0*uptilde - part->u[particle_index];
    part->v[particle_index]= 2.0*vptilde - part->v[particle_index];
    part->w[particle_index]= 2.0*wptilde - part->w[particle_index];
    part->x[particle_index] = xptilde + uptilde*dt_sub_cycling;
    part->y[particle_index] = yptilde + vptilde*dt_sub_cycling;
    part->z[particle_index] = zptilde + wptilde*dt_sub_cycling;
    
    
    //////////
    //////////
    ////////// BC
                       
    // X-DIRECTION: BC particles
    if (part->x[particle_index] > grd->Lx){
        if (param->PERIODICX==true){ // PERIODIC
            part->x[particle_index] = part->x[particle_index] - grd->Lx;
        } else { // REFLECTING BC
            part->u[particle_index] = -part->u[particle_index];
            part->x[particle_index] = 2*grd->Lx - part->x[particle_index];
        }
    }

    if (part->x[particle_index] < 0){
        if (param->PERIODICX==true){ // PERIODIC
           part->x[particle_index] = part->x[particle_index] + grd->Lx;
        } else { // REFLECTING BC
            part->u[particle_index] = -part->u[particle_index];
            part->x[particle_index] = -part->x[particle_index];
        }
    }
    
    // Y-DIRECTION: BC particles
    if (part->y[particle_index] > grd->Ly){
        if (param->PERIODICY==true){ // PERIODIC
            part->y[particle_index] = part->y[particle_index] - grd->Ly;
        } else { // REFLECTING BC
            part->v[particle_index] = -part->v[particle_index];
            part->y[particle_index] = 2*grd->Ly - part->y[particle_index];
        }
    }
    
    if (part->y[particle_index] < 0){
        if (param->PERIODICY==true){ // PERIODIC
            part->y[particle_index] = part->y[particle_index] + grd->Ly;
        } else { // REFLECTING BC
            part->v[particle_index] = -part->v[particle_index];
            part->y[particle_index] = -part->y[particle_index];
        }
    }

    // Z-DIRECTION: BC particles
    if (part->z[particle_index] > grd->Lz){
        if (param->PERIODICZ==true){ // PERIODIC
            part->z[particle_index] = part->z[particle_index] - grd->Lz;
        } else { // REFLECTING BC
            part->w[particle_index] = -part->w[particle_index];
            part->z[particle_index] = 2*grd->Lz - part->z[particle_index];
        }
    }

    if (part->z[particle_index] < 0){
        if (param->PERIODICZ==true){ // PERIODIC
            part->z[particle_index] = part->z[particle_index] + grd->Lz;
        } else { // REFLECTING BC
            part->w[particle_index] = -part->w[particle_index];
            part->z[particle_index] = -part->z[particle_index];
        }
    }
}

__global__ void gpu_mover_PC(particles* parts, EMfield* field, grid* grd, parameters* param) {
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;  // Particle number
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;  // Type of particle
    
    particles *part = &(parts[index_y]);
    if (index_x >= part->nop) {
        return;
    }

    subcycle_single_particle(part, field, grd, param, index_x);
}


void gpu_mover_PC_wrapper(particles* parts, EMfield* field, grid* grd, parameters* param, int largestNumParticles) {
    gpu_mover_PC<<<dim3(largestNumParticles / TpBx + 1, 1, 1), dim3(TpBx, param->ns, 1)>>>(parts, field, grd, param);
}

/** particle mover */
int mover_PC(struct particles* part, struct EMfield* field, struct grid* grd, struct parameters* param)
{
    // print species and subcycling
    std::cout << "***  MOVER with SUBCYCLYING "<< param->n_sub_cycles << " - species " << part->species_ID << " ***" << std::endl;
    
    // start subcycling
    for (int i_sub=0; i_sub <  part->n_sub_cycles; i_sub++){
        // move each particle with new fields
        for (int i=0; i <  part->nop; i++){
            //subcycle_single_particle(part, field, grd, param, i);                                                            
        }  // end of subcycling
    } // end of one particle

    return(0); // exit succcesfully
} // end of the mover


__device__ void interpolate_single_particle(particles* part,interpDensSpecies* ids, grid* grd, int particle_index) {

    // index of the cell
    int ix, iy, iz;

    // arrays needed for interpolation
    FPpart weight[2][2][2];
    FPpart temp[2][2][2];
    FPpart xi[2], eta[2], zeta[2];

    // determine cell: can we change to int()? is it faster?
    ix = 2 + int (floor((part->x[particle_index] - grd->xStart) * grd->invdx));
    iy = 2 + int (floor((part->y[particle_index] - grd->yStart) * grd->invdy));
    iz = 2 + int (floor((part->z[particle_index] - grd->zStart) * grd->invdz));

    // distances from node
    long xi0_index_flat = get_idx(ix - 1, iy, iz, grd->nyn, grd->nzn);
    xi[0]   = part->x[particle_index] - grd->XN_flat[xi0_index_flat];

    long eta0_index_flat = get_idx(ix, iy - 1, iz, grd->nyn, grd->nzn);
    eta[0]  = part->y[particle_index] - grd->YN_flat[eta0_index_flat];

    long zeta0_index_flat = get_idx(ix, iy, iz - 1, grd->nyn, grd->nzn);
    zeta[0] = part->z[particle_index] - grd->ZN_flat[zeta0_index_flat];

    long index_flat_1 = get_idx(ix, iy, iz, grd->nyn, grd->nzn);
    xi[1]   = grd->XN_flat[index_flat_1] - part->x[particle_index];
    eta[1]  = grd->YN_flat[index_flat_1] - part->y[particle_index];
    zeta[1] = grd->ZN_flat[index_flat_1] - part->z[particle_index];

    // calculate the weights for different nodes
    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++)
                weight[ii][jj][kk] = part->q[particle_index] * xi[ii] * eta[jj] * zeta[kk] * grd->invVOL;

    //////////////////////////
    // add charge density
    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++) {
                long rhon_index_flat = get_idx(ix - ii, iy - jj, iz - kk, grd->nyn, grd->nzn);
                ids->rhon_flat[rhon_index_flat] += weight[ii][jj][kk] * grd->invVOL;
            }


    ////////////////////////////
    // add current density - Jx
    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++)
                temp[ii][jj][kk] = part->u[particle_index] * weight[ii][jj][kk];

    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++) {
                long jx_index_flat = get_idx(ix - ii, iy - jj, iz - kk, grd->nyn, grd->nzn);
                ids->Jx_flat[jx_index_flat] += temp[ii][jj][kk] * grd->invVOL;
            }

    ////////////////////////////
    // add current density - Jy
    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++) {
                temp[ii][jj][kk] = part->v[particle_index] * weight[ii][jj][kk];
            }

    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++) {
                long jy_index_flat = get_idx(ix - ii, iy - jj, iz - kk, grd->nyn, grd->nzn);
                ids->Jy_flat[jy_index_flat] += temp[ii][jj][kk] * grd->invVOL;
            }


    ////////////////////////////
    // add current density - Jz
    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++) {
                temp[ii][jj][kk] = part->w[particle_index] * weight[ii][jj][kk];
            }

    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++) {
                long jz_index_flat = get_idx(ix - ii, iy - jj, iz - kk, grd->nyn, grd->nzn);
                ids->Jz_flat[jz_index_flat] += temp[ii][jj][kk] * grd->invVOL;
            }

    ////////////////////////////
    // add pressure pxx
    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++)
                temp[ii][jj][kk] = part->u[particle_index] * part->u[particle_index] * weight[ii][jj][kk];

    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++) {
                long pxx_index_flat = get_idx(ix - ii, iy - jj, iz - kk, grd->nyn, grd->nzn);
                ids->pxx_flat[pxx_index_flat] += temp[ii][jj][kk] * grd->invVOL;
            }


    ////////////////////////////
    // add pressure pxy
    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++)
                temp[ii][jj][kk] = part->u[particle_index] * part->v[particle_index] * weight[ii][jj][kk];

    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++) {
                long pxx_index_flat = get_idx(ix - ii, iy - jj, iz - kk, grd->nyn, grd->nzn);
                ids->pxy_flat[pxx_index_flat] += temp[ii][jj][kk] * grd->invVOL;
            }


    /////////////////////////////
    // add pressure pxz
    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++)
                temp[ii][jj][kk] = part->u[particle_index] * part->w[particle_index] * weight[ii][jj][kk];

    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++) {
                long pxz_index_flat = get_idx(ix - ii, iy - jj, iz - kk, grd->nyn, grd->nzn);
                ids->pxz_flat[pxz_index_flat] += temp[ii][jj][kk] * grd->invVOL;
            }

    /////////////////////////////
    // add pressure pyy
    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++)
                temp[ii][jj][kk] = part->v[particle_index] * part->v[particle_index] * weight[ii][jj][kk];

    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++) {
                long pyy_index_flat = get_idx(ix - ii, iy - jj, iz - kk, grd->nyn, grd->nzn);
                ids->pyy_flat[pyy_index_flat] += temp[ii][jj][kk] * grd->invVOL;
            }

    /////////////////////////////
    // add pressure pyz
    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++)
                temp[ii][jj][kk] = part->v[particle_index] * part->w[particle_index] * weight[ii][jj][kk];
                
    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++) {
                long pyz_index_flat = get_idx(ix - ii, iy - jj, iz - kk, grd->nyn, grd->nzn);
                ids->pyz_flat[pyz_index_flat] += temp[ii][jj][kk] * grd->invVOL;
            }

    /////////////////////////////
    // add pressure pzz
    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++)
                temp[ii][jj][kk] = part->w[particle_index] * part->w[particle_index] * weight[ii][jj][kk];

    for (int ii = 0; ii < 2; ii++)
        for (int jj = 0; jj < 2; jj++)
            for (int kk = 0; kk < 2; kk++) {
                long pzz_index_flat = get_idx(ix - ii, iy - jj, iz - kk, grd->nyn, grd->nzn);
                ids->pzz_flat[pzz_index_flat] += temp[ii][jj][kk] * grd->invVOL;
            }

}

__global__ void gpu_interpP2G(particles* parts, interpDensSpecies* ids, grid* grd) {
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;  // Particle number
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;  // Type of particle

    particles* part = &(parts[index_y]);
    if (index_x > part->nop) {
        return;
    }

    interpolate_single_particle(part, ids, grd, index_x);
}

void gpu_interpP2G_wrapper(particles* parts, interpDensSpecies* ids, grid* grd, parameters* param, int largestNumParticles) {
    gpu_interpP2G<<<dim3(largestNumParticles / TpBx + 1, 1, 1), dim3(TpBx, param->ns, 1)>>>(parts, ids, grd);
}

/** Interpolation Particle --> Grid: This is for species */
void interpP2G(particles* part, interpDensSpecies* ids, grid* grd)
{    
    for (register long long i = 0; i < part->nop; i++) {
        //interpolate_single_particle(part, ids, grd, i);
    }
}
